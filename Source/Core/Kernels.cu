#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include ""
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <helper_cuda_gl.h>

#include "Geometry.h"
#include "Camera.h"
#include "Utility/MathUtility.h"

// Used to convert color to a format that OpenGL can display
// Represents the color in memory as either 1 float or 4 chars (32 bits)
union HColor
{
	float Value;
	uchar4 Components;
};

namespace HKernels
{

	//////////////////////////////////////////////////////////////////////////
	// Device Kernels
	//////////////////////////////////////////////////////////////////////////
	__device__ unsigned int TWHash(unsigned int s)
	{
		s = (s ^ 61) ^ (s >> 16);
		s = s + (s << 3);
		s = s ^ (s >> 4);
		s = s * 0x27d4eb2d;
		s = s ^ (s >> 15);
		return s;
	}

	//////////////////////////////////////////////////////////////////////////
	// Global Kernels
	//////////////////////////////////////////////////////////////////////////
	__global__ void TestRenderKernel(
		float3* Pixels,
		float3* AccumulationBuffer,
		HCameraData* CameraData,
		unsigned int PassCounter)
	{
		
		int x = blockIdx.x*blockDim.x + threadIdx.x;
		int y = blockIdx.y*blockDim.y + threadIdx.y;

		// Global thread ID, used to perturb the random number generator seed
		int threadId = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

		int i = (CameraData->Resolution.y - y - 1)*CameraData->Resolution.x + x;

		// Initialize random number generator
		hiprandState RNGState;
		hiprand_init(TWHash(PassCounter) + threadId, 0, 0, &RNGState);

		// Random test color
		float3 TempColor = make_float3(hiprand_uniform(&RNGState), hiprand_uniform(&RNGState), hiprand_uniform(&RNGState));

		// Accumulate and average the color for each pass
		AccumulationBuffer[i] = (AccumulationBuffer[i] * (PassCounter - 1) + TempColor) / PassCounter;

		TempColor = AccumulationBuffer[i];

		// Make type conversion for OpenGL and perform gamma correction
		HColor Color;
		Color.Components = make_uchar4(
			(unsigned char)(powf(TempColor.x, 1 / 2.2f) * 255),
			(unsigned char)(powf(TempColor.y, 1 / 2.2f) * 255),
			(unsigned char)(powf(TempColor.z, 1 / 2.2f) * 255), 1);

		// Pass pixel coordinates and pixel color in OpenGL to output buffer
		Pixels[i] = make_float3(x, y, Color.Value);

	}

	__global__ void SavePNGKernel(
		unsigned char* ColorBytes,
		float3* Pixels,
		uint2 Resolution)
	{

		int x = blockIdx.x*blockDim.x + threadIdx.x;
		int y = blockIdx.y*blockDim.y + threadIdx.y;

		int i = (Resolution.y - y - 1)*Resolution.x + x;
		
		HColor Color;
		Color.Value = Pixels[i].z;

		ColorBytes[3 * i    ] = (unsigned char)Color.Components.x;
		ColorBytes[3 * i + 1] = (unsigned char)Color.Components.y;
		ColorBytes[3 * i + 2] = (unsigned char)Color.Components.z;

	}

	//////////////////////////////////////////////////////////////////////////
	// External CUDA access launch function
	//////////////////////////////////////////////////////////////////////////
	extern "C" void LaunchRenderKernel(
		float3* Pixels,
		float3* AccumulationBuffer,
		HCameraData* CameraData,
		HCameraData* GPUCameraData,
		unsigned int PassCounter)
	{

		const dim3 BlockSize(16, 16, 1);
		const dim3 GridSize(CameraData->Resolution.x / BlockSize.x, CameraData->Resolution.y / BlockSize.y, 1);

		TestRenderKernel<<<GridSize, BlockSize>>>(
			Pixels,
			AccumulationBuffer,
			GPUCameraData,
			PassCounter);

	}

	extern "C" void LaunchSavePNGKernel(
		unsigned char* ColorBytes,
		float3* Pixels,
		uint2 Resolution)
	{

		const dim3 BlockSize(16, 16, 1);
		const dim3 GridSize(Resolution.x / BlockSize.x, Resolution.y / BlockSize.y, 1);

		checkCudaErrors(hipDeviceSynchronize());
		SavePNGKernel<<<GridSize, BlockSize>>>(
			ColorBytes,
			Pixels,
			Resolution);
		checkCudaErrors(hipDeviceSynchronize());

	}

}
